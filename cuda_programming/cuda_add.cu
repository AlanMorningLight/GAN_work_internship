
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__
void add(unsigned long long int n,  float *x, float *y)
{
	int index = threadIdx.x;
	int stride = blockDim.x;
	for(unsigned long long int i = index; i<n; i+= stride)
		y[i]  = x[i]+ y[i];
}


int main(void)
{
		


	unsigned long long int N= 1<<29;
	float *x ,  *y;
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));

	for(unsigned long long int i = 0; i<N; i++)
	{
		x[i] = 1.0f;
		y[i] = 2.0f;

	}

	add<<<1, 256>>>(N, x, y);

	hipDeviceSynchronize();

	hipFree(x);
	hipFree(y);
	return 0;
}
